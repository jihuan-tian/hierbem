#include "hip/hip_runtime.h"
#include <deal.II/base/logstream.h>

#include <hip/hip_runtime.h>
#include <openblas-pthread/cblas.h>

#include <fstream>
#include <iostream>

#include "debug_tools.hcu"
#include "hbem_test_config.h"
#include "laplace_bem.h"

using namespace dealii;
using namespace HierBEM;

// Dirichlet boundary conditions on the left and top surface of the L-shape
class DirichletBC : public Function<3>
{
public:
  double
  value(const Point<3> &p, const unsigned int component = 0) const
  {
    (void)component;

    if (p(0) <= 1e-6)
      {
        // left surface
        return 0.0;
      }
    else
      {
        // top surface
        return 10.0;
      }
  }
};

// Neumann boundary conditions on the other surfaces of the L-shape
class NeumannBC : public Function<3>
{
public:
  double
  value(const Point<3> &p, const unsigned int component = 0) const
  {
    (void)component;
    (void)p;

    return 0;
  }
};

namespace HierBEM
{
  namespace CUDAWrappers
  {
    extern hipDeviceProp_t device_properties;
  }
} // namespace HierBEM

void
run_mixed_l_shape()
{
  // Write run-time logs to file
  std::ofstream ofs("mixed-l-shape.log");
  deallog.pop();
  deallog.depth_console(0);
  deallog.depth_file(5);
  deallog.attach(ofs);

  LogStream::Prefix prefix_string("HierBEM");

  /**
   * @internal Create and start the timer.
   */
  Timer timer;

  /**
   * @internal Set number of threads used for OpenBLAS.
   */
  openblas_set_num_threads(1);

  const size_t stack_size = 1024 * 10;
  AssertCuda(hipDeviceSetLimit(hipLimitStackSize, stack_size));
  deallog << "CUDA stack size has been set to " << stack_size << std::endl;

  /**
   * @internal Get GPU device properties.
   */
  AssertCuda(
    hipGetDeviceProperties(&HierBEM::CUDAWrappers::device_properties, 0));

  const unsigned int dim                 = 2;
  const unsigned int spacedim            = 3;
  const bool         is_interior_problem = true;

  LaplaceBEM<dim, spacedim> bem(
    1, // fe order for dirichlet space
    0, // fe order for neumann space
    1, // mapping order for dirichlet domain
    1, // mapping order for neumann domain
    LaplaceBEM<dim, spacedim>::ProblemType::MixedBCProblem,
    is_interior_problem,         // is interior problem
    4,                           // n_min for cluster tree
    32,                          // n_min for block cluster tree
    0.8,                         // eta for H-matrix
    5,                           // max rank for H-matrix
    0.01,                        // aca epsilon for H-matrix
    1.0,                         // eta for preconditioner
    2,                           // max rank for preconditioner
    0.1,                         // aca epsilon for preconditioner
    MultithreadInfo::n_threads() // Number of threads used for ACA
  );
  bem.set_project_name("mixed-l-shape");

  bem.set_dirichlet_boundary_ids({1, 2});
  bem.set_neumann_boundary_ids({19, 20, 21, 22, 23, 24});

  timer.stop();
  print_wall_time(deallog, timer, "program preparation");

  timer.start();

  bem.read_volume_mesh(HBEM_TEST_MODEL_DIR "l-shape.msh");

  timer.stop();
  print_wall_time(deallog, timer, "read mesh");

  timer.start();

  DirichletBC dirichlet_bc;
  NeumannBC   neumann_bc;

  bem.assign_dirichlet_bc(dirichlet_bc);
  bem.assign_neumann_bc(neumann_bc);

  timer.stop();
  print_wall_time(deallog, timer, "assign boundary conditions");

  timer.start();

  bem.run();

  timer.stop();
  print_wall_time(deallog, timer, "run the solver");

  deallog << "Program exits with a total wall time " << timer.wall_time() << "s"
          << std::endl;
}
