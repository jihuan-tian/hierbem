#include "hip/hip_runtime.h"
#include <deal.II/base/logstream.h>

#include <fstream>
#include <iostream>

#include "hbem_test_config.h"
#include "laplace_bem.h"

using namespace dealii;
using namespace HierBEM;

/**
 * Function object for the Neumann boundary condition data, which is also
 * the solution of the Dirichlet problem. The analytical expression is
 * \f[
 * \frac{\pdiff u}{\pdiff n}\Big\vert_{\Gamma} = \frac{\langle x-x_c,x_0-x
 * \rangle}{4\pi\norm{x_0-x}^3\rho}
 * \f]
 */
class NeumannBC : public Function<3>
{
public:
  // N.B. This function should be defined outside class NeumannBC and
  // class Example2, if not inline.
  NeumannBC()
    : Function<3>()
    , x0(0.25, 0.25, 0.25)
    , model_sphere_center(0.0, 0.0, 0.0)
    , model_sphere_radius(1.0)
  {}

  NeumannBC(const Point<3> &x0, const Point<3> &center, double radius)
    : Function<3>()
    , x0(x0)
    , model_sphere_center(center)
    , model_sphere_radius(radius)
  {}

  double
  value(const Point<3> &p, const unsigned int component = 0) const
  {
    (void)component;

    Tensor<1, 3> diff_vector = x0 - p;

    return ((p - model_sphere_center) * diff_vector) / 4.0 / numbers::PI /
           std::pow(diff_vector.norm(), 3) / model_sphere_radius;
  }

private:
  /**
   * Location of the Dirac point source \f$\delta(x-x_0)\f$.
   */
  Point<3> x0;
  Point<3> model_sphere_center;
  double   model_sphere_radius;
};

void
run_neumann_full_matrix()
{
  // Write run-time logs to file
  std::ofstream ofs("neumann-full-matrix.log");
  deallog.pop();
  deallog.depth_console(0);
  deallog.depth_file(5);
  deallog.attach(ofs);

  const unsigned int dim      = 2;
  const unsigned int spacedim = 3;

  const bool                is_interior_problem = false;
  LaplaceBEM<dim, spacedim> bem(
    1,
    0,
    1,
    1,
    LaplaceBEM<dim, spacedim>::ProblemType::NeumannBCProblem,
    is_interior_problem,
    MultithreadInfo::n_threads());
  bem.set_project_name("neumann-full-matrix");

  /**
   * @internal Set the Dirac source location according to interior or exterior
   * problem.
   */
  Point<spacedim> source_loc;

  if (is_interior_problem)
    {
      source_loc = Point<spacedim>(1, 1, 1);
    }
  else
    {
      source_loc = Point<spacedim>(0.25, 0.25, 0.25);
    }

  const Point<spacedim> center(0, 0, 0);
  const double          radius(1);

  Triangulation<spacedim> tria;
  // The manifold_id is set to 0 on the boundary faces in @p hyper_ball.
  GridGenerator::hyper_ball(tria, center, radius);
  tria.refine_global(1);

  bem.assign_volume_triangulation(std::move(tria), true);

  Triangulation<dim, spacedim>           surface_tria;
  const SphericalManifold<dim, spacedim> ball_surface_manifold(center);
  surface_tria.set_manifold(0, ball_surface_manifold);

  bem.assign_surface_triangulation(std::move(surface_tria), true);

  NeumannBC neumann_bc(source_loc, center, radius);
  bem.assign_neumann_bc(neumann_bc);

  bem.run();
}
