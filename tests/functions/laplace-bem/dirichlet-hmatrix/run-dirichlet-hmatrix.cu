#include "hip/hip_runtime.h"
#include <deal.II/base/logstream.h>

#include <hip/hip_runtime.h>
#include <openblas-pthread/cblas.h>

#include <fstream>
#include <iostream>

#include "cu_profile.hcu"
#include "debug_tools.hcu"
#include "hbem_test_config.h"
#include "laplace_bem.h"

using namespace dealii;
using namespace HierBEM;

/**
 * Function object for the Dirichlet boundary condition data, which is
 * also the solution of the Neumann problem. The analytical expression is:
 * \f[
 * u=\frac{1}{4\pi\norm{x-x_0}}
 * \f]
 */
class DirichletBC : public Function<3>
{
public:
  // N.B. This function should be defined outside class NeumannBC or class
  // Example2, if no inline.
  DirichletBC()
    : Function<3>()
    , x0(0.25, 0.25, 0.25)
  {}

  DirichletBC(const Point<3> &x0)
    : Function<3>()
    , x0(x0)
  {}

  double
  value(const Point<3> &p, const unsigned int component = 0) const
  {
    (void)component;
    return 1.0 / 4.0 / numbers::PI / (p - x0).norm();
  }

private:
  /**
   * Location of the Dirac point source \f$\delta(x-x_0)\f$.
   */
  Point<3> x0;
};


namespace HierBEM
{
  namespace CUDAWrappers
  {
    extern hipDeviceProp_t device_properties;
  }
} // namespace HierBEM

void
run_dirichlet_hmatrix()
{
  /**
   * @internal Pop out the default "DEAL" prefix string.
   */
  // Write run-time logs to file
  std::ofstream ofs("dirichlet-hmatrix.log");
  deallog.pop();
  deallog.depth_console(0);
  deallog.depth_file(5);
  deallog.attach(ofs);

  LogStream::Prefix prefix_string("HierBEM");
#if ENABLE_NVTX == 1
  HierBEM::CUDAWrappers::NVTXRange nvtx_range("HierBEM");
#endif

  /**
   * @internal Create and start the timer.
   */
  Timer timer;

  /**
   * @internal Set number of threads used for OpenBLAS.
   */
  openblas_set_num_threads(1);

  /**
   * @internal Initialize the CUDA device parameters.
   */
  //  AssertCuda(hipSetDevice(0));
  //  AssertCuda(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceScheduleBlockingSync));

  const size_t stack_size = 1024 * 10;
  AssertCuda(hipDeviceSetLimit(hipLimitStackSize, stack_size));
  deallog << "CUDA stack size has been set to " << stack_size << std::endl;

  /**
   * @internal Get GPU device properties.
   */
  AssertCuda(
    hipGetDeviceProperties(&HierBEM::CUDAWrappers::device_properties, 0));

  /**
   * @internal Use 8-byte bank size in shared memory, since double value type is
   * used.
   */
  // AssertCuda(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  const unsigned int dim      = 2;
  const unsigned int spacedim = 3;

  const bool                is_interior_problem = true;
  LaplaceBEM<dim, spacedim> bem(
    1, // fe order for dirichlet space
    0, // fe order for neumann space
    1, // mapping order for dirichlet domain
    1, // mapping order for neumann domain
    LaplaceBEM<dim, spacedim>::ProblemType::DirichletBCProblem,
    is_interior_problem,         // is interior problem
    4,                           // n_min for cluster tree
    4,                           // n_min for block cluster tree
    0.8,                         // eta for H-matrix
    5,                           // max rank for H-matrix
    0.01,                        // aca epsilon for H-matrix
    1.0,                         // eta for preconditioner
    2,                           // max rank for preconditioner
    0.1,                         // aca epsilon for preconditioner
    MultithreadInfo::n_threads() // Number of threads used for ACA
  );
  bem.set_project_name("dirichlet-hmatrix");

  timer.stop();
  print_wall_time(deallog, timer, "program preparation");

  timer.start();

  /**
   * @internal Set the Dirac source location according to interior or exterior
   * problem.
   */
  Point<spacedim> source_loc;

  if (is_interior_problem)
    {
      source_loc = Point<spacedim>(1, 1, 1);
    }
  else
    {
      source_loc = Point<spacedim>(0.25, 0.25, 0.25);
    }

  const Point<spacedim> center(0, 0, 0);
  const double          radius(1);

  Triangulation<spacedim> tria;
  // The manifold_id is set to 0 on the boundary faces in @p hyper_ball.
  GridGenerator::hyper_ball(tria, center, radius);
  tria.refine_global(1);

  bem.assign_volume_triangulation(std::move(tria), true);

  Triangulation<dim, spacedim>           surface_tria;
  const SphericalManifold<dim, spacedim> ball_surface_manifold(center);
  surface_tria.set_manifold(0, ball_surface_manifold);

  bem.assign_surface_triangulation(std::move(surface_tria), true);

  timer.stop();
  print_wall_time(deallog, timer, "read mesh");

  timer.start();

  DirichletBC dirichlet_bc(source_loc);
  bem.assign_dirichlet_bc(dirichlet_bc);

  timer.stop();
  print_wall_time(deallog, timer, "assign boundary conditions");

  timer.start();

  bem.run();

  timer.stop();
  print_wall_time(deallog, timer, "run the solver");

  deallog << "Program exits with a total wall time " << timer.wall_time() << "s"
          << std::endl;

  bem.print_memory_consumption_table(deallog.get_file_stream());
}
