#include "hip/hip_runtime.h"
/**
 * \file laplace-bem-dirichlet-hmatrix.cc
 * \brief
 * \ingroup testers
 * \author Jihuan Tian
 * \date 2022-09-23
 */

#include <deal.II/base/logstream.h>

#include <hip/hip_runtime.h>

#include <iostream>

#include "cu_profile.hcu"
#include "debug_tools.hcu"
#include "hbem_test_config.h"
#include "laplace_bem.h"

using namespace dealii;
using namespace HierBEM;

/**
 * Function object for the Dirichlet boundary condition data, which is
 * also the solution of the Neumann problem. The analytical expression is:
 * \f[
 * u=\frac{1}{4\pi\norm{x-x_0}}
 * \f]
 */
class DirichletBC : public Function<3>
{
public:
  // N.B. This function should be defined outside class NeumannBC or class
  // Example2, if no inline.
  DirichletBC()
    : Function<3>()
    , x0(0.25, 0.25, 0.25)
  {}

  DirichletBC(const Point<3> &x0)
    : Function<3>()
    , x0(x0)
  {}

  double
  value(const Point<3> &p, const unsigned int component = 0) const
  {
    (void)component;
    return 1.0 / 4.0 / numbers::PI / (p - x0).norm();
  }

private:
  /**
   * Location of the Dirac point source \f$\delta(x-x_0)\f$.
   */
  Point<3> x0;
};

/**
 * Function object for the Neumann boundary condition data, which is also
 * the solution of the Dirichlet problem. The analytical expression is
 * \f[
 * \frac{\pdiff u}{\pdiff n}\Big\vert_{\Gamma} = \frac{\langle x-x_c,x_0-x
 * \rangle}{4\pi\norm{x_0-x}^3\rho}
 * \f]
 */
class NeumannBC : public Function<3>
{
public:
  // N.B. This function should be defined outside class NeumannBC and
  // class Example2, if not inline.
  NeumannBC()
    : Function<3>()
    , x0(0.25, 0.25, 0.25)
    , model_sphere_center(0.0, 0.0, 0.0)
    , model_sphere_radius(1.0)
  {}

  NeumannBC(const Point<3> &x0, const Point<3> &center, double radius)
    : Function<3>()
    , x0(x0)
    , model_sphere_center(center)
    , model_sphere_radius(radius)
  {}

  double
  value(const Point<3> &p, const unsigned int component = 0) const
  {
    (void)component;

    Tensor<1, 3> diff_vector = x0 - p;

    return ((p - model_sphere_center) * diff_vector) / 4.0 / numbers::PI /
           std::pow(diff_vector.norm(), 3) / model_sphere_radius;
  }

private:
  /**
   * Location of the Dirac point source \f$\delta(x-x_0)\f$.
   */
  Point<3> x0;
  Point<3> model_sphere_center;
  double   model_sphere_radius;
};

namespace HierBEM
{
  namespace CUDAWrappers
  {
    extern hipDeviceProp_t device_properties;
  }
} // namespace HierBEM

int
main(int argc, char *argv[])
{
  /**
   * @internal Pop out the default "DEAL" prefix string.
   */
  deallog.pop();
  deallog.depth_console(5);
  LogStream::Prefix prefix_string("HierBEM");
#if ENABLE_NVTX == 1
  HierBEM::CUDAWrappers::NVTXRange nvtx_range("HierBEM");
#endif

  /**
   * @internal Create and start the timer.
   */
  Timer timer;

  /**
   * @internal Initialize the CUDA device parameters.
   */
  //  AssertCuda(hipSetDevice(0));
  //  AssertCuda(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceScheduleBlockingSync));

  const size_t stack_size = 1024 * 10;
  AssertCuda(hipDeviceSetLimit(hipLimitStackSize, stack_size));
  deallog << "CUDA stack size has been set to " << stack_size << std::endl;

  /**
   * @internal Get GPU device properties.
   */
  AssertCuda(
    hipGetDeviceProperties(&HierBEM::CUDAWrappers::device_properties, 0));

  /**
   * @internal Use 8-byte bank size in shared memory, since double value type is
   * used.
   */
  // AssertCuda(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  const unsigned int dim      = 2;
  const unsigned int spacedim = 3;

  const bool                is_interior_problem = true;
  LaplaceBEM<dim, spacedim> bem(
    1, // fe order for dirichlet space
    0, // fe order for neumann space
    1, // mapping order for dirichlet domain
    1, // mapping order for neumann domain
    LaplaceBEM<dim, spacedim>::ProblemType::DirichletBCProblem,
    is_interior_problem, // is interior problem
    4,                   // n_min for cluster tree
    10,                  // n_min for block cluster tree
    0.8,                 // eta for H-matrix
    5,                   // max rank for H-matrix
    0.01,                // aca epsilon for H-matrix
    1.0,                 // eta for preconditioner
    2,                   // max rank for preconditioner
    0.1,                 // aca epsilon for preconditioner
    MultithreadInfo::n_cores());

  timer.stop();
  print_wall_time(deallog, timer, "program preparation");

  timer.start();

  if (argc > 1)
    {
      bem.read_volume_mesh(argv[1]);
    }
  else
    {
      bem.read_volume_mesh(HBEM_TEST_MODEL_DIR "sphere.msh");
    }

  timer.stop();
  print_wall_time(deallog, timer, "read mesh");

  timer.start();

  /**
   * @internal Set the Dirac source location according to interior or exterior
   * problem.
   */
  Point<3> source_loc;

  if (is_interior_problem)
    {
      source_loc = Point<3>(1, 1, 1);
      // source_loc = Point<3>(1.5, 1.5, 1.5);
    }
  else
    {
      source_loc = Point<3>(0.25, 0.25, 0.25);
    }

  const Point<3> center(0, 0, 0);
  const double   radius(1);
  // const double radius(1.5);

  DirichletBC dirichlet_bc(source_loc);
  NeumannBC   neumann_bc(source_loc, center, radius);

  bem.assign_dirichlet_bc(dirichlet_bc);
  bem.assign_neumann_bc(neumann_bc);

  timer.stop();
  print_wall_time(deallog, timer, "assign boundary conditions");

  timer.start();

  bem.run();

  timer.stop();
  print_wall_time(deallog, timer, "run the solver");

  deallog << "Program exits with a total wall time " << timer.wall_time() << "s"
          << std::endl;

  return 0;
}
