/**
 * @file verify-cuda-table.cu
 * @brief Verify the implementation of @p CUDATable class, whose memory is
 * managed from the host.
 *
 * @ingroup testers
 * @author Jihuan Tian
 * @date 2023-01-16
 */

#include <iostream>
#include <vector>

#include "cu_table.hcu"


using namespace HierBEM;
using namespace dealii;

int
main()
{
  {
    std::cout << "=== Synchronous allocation and releasing" << std::endl;

    std::vector<double> values{1, 2, 3, 4, 5, 6, 7, 8};

    TableIndices<3>  indices(2, 2, 2);
    Table<3, double> table_cpu(2, 2, 2, values.begin());

    CUDAWrappers::CUDATable<3, double> table_gpu1;
    CUDAWrappers::CUDATable<3, double> table_gpu2;
    table_gpu1.allocate(indices);
    table_gpu1.assign_from_host(table_cpu);
    std::cout << "Table1:" << std::endl;
    print_cuda_object<<<1, 10>>>(table_gpu1);
    hipDeviceSynchronize();

    table_gpu2 = table_gpu1;
    std::cout << "Table2:" << std::endl;
    print_cuda_object<<<1, 10>>>(table_gpu2);
    hipDeviceSynchronize();

    table_gpu1.release();
    table_gpu2.release();
  }

  {
    std::cout << "=== Asynchronous allocation and releasing" << std::endl;

    hipStream_t stream;
    hipStreamCreate(&stream);

    std::vector<double> values{1, 2, 3, 4, 5, 6, 7, 8};

    TableIndices<3>  indices(2, 2, 2);
    Table<3, double> table_cpu(2, 2, 2, values.begin());

    CUDAWrappers::CUDATable<3, double> table_gpu1;
    CUDAWrappers::CUDATable<3, double> table_gpu2;

    table_gpu1.allocate(indices, stream);
    table_gpu1.assign_from_host(table_cpu, stream);
    hipStreamSynchronize(stream);

    std::cout << "Table1:" << std::endl;
    print_cuda_object<<<1, 10>>>(table_gpu1);
    hipDeviceSynchronize();

    table_gpu2.assign(table_gpu1, stream);
    hipStreamSynchronize(stream);
    std::cout << "Table2:" << std::endl;
    print_cuda_object<<<1, 10>>>(table_gpu2);
    hipDeviceSynchronize();

    table_gpu1.release(stream);
    table_gpu2.release(stream);
    hipStreamSynchronize(stream);

    hipStreamDestroy(stream);
  }

  return 0;
}
