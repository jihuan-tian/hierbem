#include "hip/hip_runtime.h"
#include <catch2/catch_all.hpp>

#include <fstream>

#include "hmatrix/hmatrix.h"
#include "linear_algebra/lapack_full_matrix_ext.h"
#include "utilities/read_octave_data.h"

using namespace HierBEM;
using namespace Catch::Matchers;

void
run_hmatrix_symm_vmult_symm()
{
  std::ofstream ofs("hmatrix-symm-vmult-symm.output");

  LAPACKFullMatrixExt<double> M;
  std::ifstream               in("M.dat");
  M.read_from_mat(in, "M");
  in.close();
  REQUIRE(M.size()[0] > 0);
  REQUIRE(M.size()[0] == M.size()[1]);

  /**
   * Set the property of the full matrix as @p symmetric.
   */
  M.set_property(LAPACKSupport::symmetric);

  /**
   * Read the vector \f$x\f$.
   */
  Vector<double> x;
  in.open("x.dat");
  read_vector_from_octave(in, "x", x);
  in.close();
  REQUIRE(x.size() == M.size()[0]);

  /**
   * Generate index set.
   */
  const unsigned int                   p = 6;
  const unsigned int                   n = std::pow(2, p);
  std::vector<types::global_dof_index> index_set(n);

  for (unsigned int i = 0; i < n; i++)
    {
      index_set.at(i) = i;
    }

  const unsigned int n_min = 2;

  /**
   * Generate cluster tree.
   */
  ClusterTree<3> cluster_tree(index_set, n_min);
  cluster_tree.partition();

  /**
   * Generate block cluster tree with the two component cluster trees being the
   * same.
   */
  BlockClusterTree<3, double> block_cluster_tree(cluster_tree, cluster_tree);
  block_cluster_tree.partition_fine_non_tensor_product();

  /**
   * Generate the \hmatrix from the symmetric full matrix. Its property will
   * be automatically set to @p HMatrixSupport::Property::symmetric.
   */
  const unsigned int fixed_rank_k = n / 4;
  HMatrix<3, double> H(block_cluster_tree, M, fixed_rank_k);
  REQUIRE(H.get_m() == M.size()[0]);
  REQUIRE(H.get_n() == M.size()[1]);

  LAPACKFullMatrixExt<double> H_full;
  H.convertToFullMatrix(H_full);
  REQUIRE(H_full.size()[0] == M.size()[0]);
  REQUIRE(H_full.size()[1] == M.size()[1]);

  H_full.print_formatted_to_mat(ofs, "H_full", 15, false, 25, "0");

  /**
   * Perform matrix/vector multiplication.

   * \alert{When the \hmatrix is symmetric, its symmetry property should be
   * passed as the third argument to @p vmult.}
   */
  Vector<double> y(n);
  H.vmult(y, x);
  print_vector_to_mat(ofs, "y1", y, false);

  H.vmult(y, 0.5, x);
  print_vector_to_mat(ofs, "y2", y, false);

  ofs.close();
}
