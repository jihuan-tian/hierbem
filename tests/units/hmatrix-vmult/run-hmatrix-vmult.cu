#include "hip/hip_runtime.h"
#include <catch2/catch_all.hpp>

#include <fstream>

#include "hmatrix.h"
#include "lapack_full_matrix_ext.h"
#include "read_octave_data.h"

using namespace HierBEM;
using namespace Catch::Matchers;

void
run_hmatrix_vmult()
{
  std::ofstream ofs("hmatrix-vmult.output");

  /**
   * Load a general matrix.
   */
  LAPACKFullMatrixExt<double> M;
  std::ifstream               in("M.dat");
  M.read_from_mat(in, "M");
  in.close();
  REQUIRE(M.size()[0] > 0);
  REQUIRE(M.size()[0] == M.size()[1]);

  /**
   * Generate index set.
   */
  const unsigned int                   p = 6;
  const unsigned int                   n = std::pow(2, p);
  std::vector<types::global_dof_index> index_set(n);

  for (unsigned int i = 0; i < n; i++)
    {
      index_set.at(i) = i;
    }

  /**
   * Generate cluster tree.
   */
  const unsigned int n_min = 2;
  ClusterTree<3>     cluster_tree(index_set, n_min);
  cluster_tree.partition();

  /**
   * Generate block cluster tree with the two component cluster trees being the
   * same.
   */
  BlockClusterTree<3, double> block_cluster_tree(cluster_tree, cluster_tree);
  block_cluster_tree.partition_fine_non_tensor_product();

  /**
   * Create a rank-k HMatrix.
   */
  const unsigned int fixed_rank_k = n / 4;
  HMatrix<3, double> H(block_cluster_tree, M, fixed_rank_k);
  REQUIRE(H.get_m() == M.size()[0]);
  REQUIRE(H.get_n() == M.size()[1]);

  /**
   * Convert the \hmatrix back to full matrix for comparison with the original
   * full matrix.
   */
  LAPACKFullMatrixExt<double> H_full;
  H.convertToFullMatrix(H_full);
  REQUIRE(H_full.size()[0] == M.size()[0]);
  REQUIRE(H_full.size()[1] == M.size()[1]);

  H_full.print_formatted_to_mat(ofs, "H_full", 15, false, 25, "0");

  /**
   * Read the vector \f$x\f$.
   */
  Vector<double> x;
  in.open("x.dat");
  read_vector_from_octave(in, "x", x);
  in.close();
  REQUIRE(x.size() == M.size()[1]);

  /**
   * Perform \hmatrix/vector multiplication.
   */
  Vector<double> y(n);
  H.vmult(y, x);
  print_vector_to_mat(ofs, "y1", y);

  y = 0.;
  H.vmult(y, 0.5, x);
  print_vector_to_mat(ofs, "y2", y);

  ofs.close();
}
